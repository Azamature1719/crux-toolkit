#include "hip/hip_runtime.h"
#include <iostream>
#include "gpu_scoring.cuh"

int *d_peptides;

size_t pep_length;
size_t pep_num;

__global__ void score(int *d_peptides, int *d_cache, int *d_result, size_t pep_length){
  
  int current_peptide = blockDim.x * blockIdx.x + threadIdx.x;
  int result = 0;
  for (size_t i = 0; i < pep_length; ++i) {
      int peak = d_peptides[current_peptide * pep_length + i];
      if(peak != -1){
        result += d_cache[peak];
      }
  }
  d_result[current_peptide] = result;

}

void transferPeptides(std::vector<std::vector<int>> peptides){

   int devices = 0; 
   hipError_t err = hipGetDeviceCount(&devices); 

   if (devices > 0 && err == hipSuccess) 
   { 
     // Set default GPU. If there are several devices, one of them could be set
     hipSetDevice(0);
      
     // Get all free memory
     // size_t free, total;
     // hipMemGetInfo(&free, &total);

     // Form a single vector 
     std::vector<int> peptides_to_transfer;
     for(auto pep_iter = peptides.begin(); pep_iter != peptides.end(); pep_iter++){
       copy((*pep_iter).begin(), (*pep_iter).end(), back_inserter(peptides_to_transfer));
     }

     hipError_t err = hipMalloc((void **)&d_peptides, peptides_to_transfer.size() * sizeof(int));
     // std::cout << "\nPeptides malloc error: " << err;
     err = hipMemcpy(d_peptides, peptides_to_transfer.data(), peptides_to_transfer.size() * sizeof(int), hipMemcpyHostToDevice);
     // std::cout << "\nPeptides memcpy error: " << err;
  
     // Set a number of transmitted peptides and a single peptide's length
     pep_num = peptides.size();
     pep_length = peptides[0].size();
   } 
 }

 std::vector<int> applyScoring(const int *cache, unsigned int cache_size){
   
   // Configure GPU thread grid parameters. 32 is a number of threads executed in a single warp
   size_t block_size = 32;
   size_t grid_size = (pep_num + 1) / block_size + 1;

   // Allocate mem for cache peptides
   int *d_cache, *d_result;
   hipError_t err = hipMalloc((void **)&d_cache, cache_size * sizeof(int));

   // Copy cache peptides to GPU
   err = hipMemcpy(d_cache, cache, cache_size * sizeof(int), hipMemcpyHostToDevice);
   // std::cout << "\nMemcpy cache error: " << err;

   err = hipMalloc((void **)&d_result, pep_num * sizeof(int));
   // std::cout << "\nMalloc result error: " << err;
   
   // Run scoring function on GPU
   score <<<block_size, grid_size>>>(d_peptides, d_cache, d_result, pep_length);
  
   // Get results from GPU
   std::vector<int> result(pep_num);
   err = hipMemcpy(result.data(), d_result, pep_num * sizeof(int), hipMemcpyDeviceToHost);
   // std::cout << "\nMemcpy result error: " << err;

   hipFree(d_cache);
   hipFree(d_peptides);
   hipFree(d_result);

   return result;
}
