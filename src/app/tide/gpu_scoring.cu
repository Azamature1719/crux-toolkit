#include "hip/hip_runtime.h"
#include <iostream>
#include "gpu_scoring.cuh"

// int *d_peptides;
// int *d_cache;

// size_t pep_length;
// size_t pep_num;

__global__ void score(int *d_peptides, int *d_cache, int *d_result, size_t pep_length){
  
  int current_peptide = blockDim.x * blockIdx.x + threadIdx.x;
  int result = 0;
  for (size_t i = 0; i < pep_length; ++i) {
      int peak = d_peptides[current_peptide * pep_length + i];
      if(peak != -1){
        result += d_cache[peak];
      }
  }
  d_result[current_peptide] = result;

}

std::vector<int> applyScoring(std::vector<std::vector<int>> peptides, const int *cache, unsigned int cache_size){

  int devices = 0; 
  hipError_t err = hipGetDeviceCount(&devices); 

  if (devices > 0 && err == hipSuccess) 
  { 
    hipSetDevice(0);
      
    // size_t free, total;
    // hipMemGetInfo(&free, &total);

    // Form a single vector 
    std::vector<int> peptides_to_transfer;
    for(auto pep_iter = peptides.begin(); pep_iter != peptides.end(); pep_iter++){
      copy((*pep_iter).begin(), (*pep_iter).end(), back_inserter(peptides_to_transfer));
    }

    int *d_peptides;
    hipError_t err = hipMalloc((void **)&d_peptides, peptides_to_transfer.size() * sizeof(int));
   // std::cout << "\nPep allocation: " << err;
    err = hipMemcpy(d_peptides, peptides_to_transfer.data(), peptides_to_transfer.size() * sizeof(int), hipMemcpyHostToDevice);
   // std::cout << "\nPeptides memcpy: " << err;

    size_t block_size = 32;
    size_t grid_size = (peptides.size() + 1) / block_size + 1;

   // std::cout << "\nBlock_size = " << block_size << " Grid_size = " << grid_size;

    // transfer cache
    // d_cache = d_peptides + pep_num * pep_length + 1;

    int *d_cache, *d_result;
    err = hipMalloc((void **)&d_cache, cache_size * sizeof(int));
   //  std::cout << "\nCache malloc: " << err;
    err = hipMemcpy(d_cache, cache, cache_size * sizeof(int), hipMemcpyHostToDevice);
   //  std::cout << "\nCache memcpy: " << err;

    // allocate memory for result variable
    //int *d_result = d_cache + cache_size + 1;

    err = hipMalloc((void **)&d_result, peptides.size() * sizeof(int));
   //  std::cout << "\nResults malloc: " << err;

    // score
   // std::cout << "\nPepLength: " << peptides[0].size();
    score <<<grid_size, block_size>>>(d_peptides, d_cache, d_result, peptides[0].size());
    
    // get results
    // hipDeviceSynchronize();
    std::vector<int> result(peptides.size());
    int *res = new int[peptides.size()];
    err = hipMemcpy(res, d_result, peptides.size() * sizeof(int), hipMemcpyDeviceToHost);


    for(int i = 0; i < peptides.size(); ++i){
      int j = 0;
      int pep_res = 0;
      while(peptides[i][j] != -1){
        pep_res += cache[peptides[i][j]];
        j++;
      }
      std::cout << " Pep_res " << i << " :" << pep_res;
    }

    std::cout << "\n\n";

    for(int i = 0; i < peptides.size(); ++i){
      std::cout << " Res " << i << " :" << res[i];
    }

    // hipDeviceSynchronize();
    // std::cout << "\nResults got: " << err;

    hipFree(d_cache);
    hipFree(d_peptides);
    hipFree(d_result);

    return result;
  } 
}

// void transferPeptides(std::vector<std::vector<int>> peptides){

//   int devices = 0; 
//   hipError_t err = hipGetDeviceCount(&devices); 

//   if (devices > 0 && err == hipSuccess) 
//   { 
//     hipSetDevice(0);
      
//     size_t free, total;
//     hipMemGetInfo(&free, &total);

//     // Form a single vector 
//     std::vector<int> peptides_to_transfer;
//     for(auto pep_iter = peptides.begin(); pep_iter != peptides.end(); pep_iter++){
//       copy((*pep_iter).begin(), (*pep_iter).end(), back_inserter(peptides_to_transfer));
//     }

//     std::cout << "peps allocation";
//     hipMalloc((void **)&d_peptides, peptides_to_transfer.size() * sizeof(int));
//     hipMemcpy(d_peptides, peptides_to_transfer.data(), peptides_to_transfer.size() * sizeof(int), hipMemcpyHostToDevice);
//     std::cout << "peps allocated";

//     pep_num = peptides.size();
//     pep_length = peptides[0].size();
//   } 
// }

// std::vector<int> applyScoring(const int *cache, unsigned int cache_size){

//   size_t block_size = 32;
//   size_t grid_size = (pep_num + 1) / block_size + 1;

//   // transfer cache
//   //  d_cache = d_peptides + pep_num * pep_length + 1;

//   int *d_cache, *d_result;
//   hipMalloc((void **)&d_cache, cache_size * sizeof(int));
//   hipMemcpy(d_cache, cache, cache_size * sizeof(int), hipMemcpyHostToDevice);

//   // allocate memory for result variable
//   //int *d_result = d_cache + cache_size + 1;

//   hipMalloc((void **)&d_result, pep_num * sizeof(int));

//   // score
//   std::cout << "mem allocated";
//   score <<<block_size, grid_size>>>(d_peptides, d_cache, d_result, pep_length);
  
//   // get results
//   std::vector<int> result;
//   hipMemcpy(result.data(), d_result, pep_num * sizeof(int), hipMemcpyDeviceToHost);

//   std::cout << "scored";

//   hipFree(d_cache);
//   hipFree(d_peptides);
//   hipFree(d_result);

//   return result;
// }
