#include "hip/hip_runtime.h"
#include "gpu_scoring.cuh"

unsigned int *d_peptides;
unsigned int *d_res;

std::string setDeviceProperties(int deviceNum, size_t warpSize, size_t spectrumMatchingOnce, std::vector<unsigned int> peptides){
  std::string result;
  
  int devices = 0; 
  hipError_t err = hipGetDeviceCount(&devices); 

  if (devices > 0 && err == hipSuccess) 
  { 
    result += std::to_string(devices);
    hipSetDevice(deviceNum);
      
    // Get GPU device properties - could be used for memory configuration
    // hipDeviceProp_t deviceProp; 
    // hipGetDeviceProperties(&deviceProp, deviceNum);

    // Get all free memory on GPU device
    size_t free, total;

    hipMemGetInfo(&free, &total);
    result += " FREE: " + std::to_string(free);
    result += " TOTAL: " + std::to_string(total);

    err = hipMalloc((void **)&d_peptides, peptides.size() * sizeof(unsigned int));
    result += " MALLOC D_PEPTIDES: " + std::to_string(err);

    err = hipMalloc((void **)&d_res, sizeof(unsigned int));
    result += " MALLOC D_RES: " + std::to_string(err);

    // Calculating property values
    size_t block_size = warpSize;
    size_t grid_size = (spectrumMatchingOnce + 1) / block_size + 1;

    hipMemcpy(d_peptides, peptides.data(), peptides.size() * sizeof(unsigned int), hipMemcpyHostToDevice);
    score <<<1,1>>>(d_peptides, d_res, peptides.size());

  //  unsigned int *res;
  //  hipMemcpy(res, d_res, sizeof(unsigned int), hipMemcpyDeviceToHost);

  //  result += " SCORING: " + res[0];

  } 
  return result;
}

// hipError_t transferDataToDevice(std::vector<unsigned int> peptides){
//   return hipMemcpy(d_peptides, peptides.data(), peptides.size() * sizeof(unsigned int), hipMemcpyHostToDevice);
// }

// scoring function
__global__ void score(unsigned int *d_peptides, unsigned int *d_res, size_t peptides_size){
  for(size_t i = 0; i < peptides_size; ++i){
    d_res[0] += d_peptides[i];
  }
}
