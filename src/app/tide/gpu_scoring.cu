#include "hip/hip_runtime.h"
#include "gpu_scoring.cuh"

unsigned int *d_peaks;
int *d_cache;
int *d_result;

size_t peaks_size;

void transferPeaks(unsigned int deviceNum, std::vector<unsigned int> peaks){

  int devices = 0; 
  hipError_t err = hipGetDeviceCount(&devices); 

  if (devices > 0 && err == hipSuccess) 
  { 
    hipSetDevice(deviceNum);
      
    size_t free, total;
    hipMemGetInfo(&free, &total);
    hipMalloc((void **)&d_peaks, free);
    hipMemcpy(d_peaks, peaks.data(), peaks.size() * sizeof(unsigned int), hipMemcpyHostToDevice);

    peaks_size = peaks.size();
  } 
}

void transferCache(size_t warpSize, const int *cache, unsigned int size_cache){

    hipMalloc((void **)&d_cache, size_cache * sizeof(int));
    hipMemcpy(d_cache, cache, size_cache * sizeof(int), hipMemcpyHostToDevice);
}

int applyScoring(){
  
  //size_t block_size = warpSize;
  //size_t grid_size = (peptides.size() + 1) / block_size + 1;

  int result = 0;

  score <<<1,1>>>(d_peaks, d_cache, d_result, peaks_size);
  hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);
  
  return result;
}

__global__ void score(unsigned int *d_peaks, int *d_cache, int d_result, size_t peaks_size){

  d_result = 0;

  for(size_t i = 0; i < peaks_size; ++i){
    d_result += d_cache[d_peaks[i]];
  }
}
