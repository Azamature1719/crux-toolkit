#include "hip/hip_runtime.h"
#include <iostream>
#include "gpu_scoring.cuh"

unsigned int *d_peaks;
int *d_cache;
int *d_result;

size_t peaks_size;
size_t pep_num;

__global__ void score(unsigned int *d_peaks, int *d_cache, int *d_result, size_t peaks_size){
  d_result[0] = 0;
  for(size_t i = 0; i < peaks_size; ++i){
    d_result[0] += d_cache[d_peaks[i]];
  }
  
  // -- Align the peptides -- 
  
  // int id = threadIdx.x;
  // d_result[id] = 0;
  // for(size_t i = 0; i < peaks_size; ++i){
  //   d_result[id] += d_cache[d_peaks[i]];
  // }
}

void transferPeaks(unsigned int deviceNum, std::vector<unsigned int> peaks, size_t _pep_num){

  int devices = 0; 
  hipError_t err = hipGetDeviceCount(&devices); 

  std::cout << "\nPEAKS: " << "SIZE: " << peaks.size() << "\n";
  for(size_t i = 0; i < 10; ++i){
    std::cout << peaks[i] << "\n";
  }

  if (devices > 0 && err == hipSuccess) 
  { 
    hipSetDevice(deviceNum);
      
    size_t free, total;
    hipMemGetInfo(&free, &total);
    hipMalloc((void **)&d_peaks, free);
    hipMemcpy(d_peaks, peaks.data(), peaks.size() * sizeof(unsigned int), hipMemcpyHostToDevice);

    peaks_size = peaks.size();
    pep_num = _pep_num;
  } 
}

std::vector<int> applyScoring(size_t warpSize, const int *cache, unsigned int cache_size){

  std::cout << "\nCACHE: " << "SIZE: " << cache_size << "\n";
  for(size_t i = 0; i < 10; ++i){
    std::cout << cache[i] << "\n";
  }

  size_t block_size = warpSize;
  size_t grid_size = (peaks_size + 1) / block_size + 1;

  // transfer cache
  d_cache = (int*)(d_peaks + peaks_size);
  hipMalloc((void **)&d_cache, cache_size * sizeof(int));
  hipMemcpy(d_cache, cache, cache_size * sizeof(int), hipMemcpyHostToDevice);

  // allocate memory for result variable
  std::vector<int> result;

  d_result = (int*)(d_cache + cache_size);
  hipMalloc((void **)&d_result, pep_num * sizeof(int));

  // score
  score <<<block_size, grid_size>>>(d_peaks, d_cache, d_result, peaks_size);
  hipMemcpy(result.data(), d_result, pep_num * sizeof(int), hipMemcpyDeviceToHost);
  
  return result;
}
