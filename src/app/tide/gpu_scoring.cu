#include "hip/hip_runtime.h"
#include <iostream>
#include "gpu_scoring.cuh"

int *d_peptides;
//int *d_cache;

size_t pep_length;
size_t pep_num;

__global__ void score(int *d_peptides, int *d_cache, int *d_result, size_t pep_length){
  
  int current_peptide = blockDim.x * blockIdx.x + threadIdx.x;
  int result = 0;
  for (size_t i = 0; i < pep_length; ++i) {
      int peak = d_peptides[current_peptide * pep_length + i];
      if (peak != -1) {
          result += d_cache[peak];
      }
  }
  d_result[current_peptide] = result;

}

void transferPeptides(std::vector<std::vector<int>> peptides){

  int devices = 0; 
  hipError_t err = hipGetDeviceCount(&devices); 

  if (devices > 0 && err == hipSuccess) 
  { 
    hipSetDevice(0);
      
    size_t free, total;
    hipMemGetInfo(&free, &total);

    // Form a single vector 
    std::vector<int> peptides_to_transfer;
    for(auto pep_iter = peptides.begin(); pep_iter != peptides.end(); pep_iter++){
      copy((*pep_iter).begin(), (*pep_iter).end(), back_inserter(peptides_to_transfer));
    }

    std::cout << "peps allocation";
    hipMalloc((void **)&d_peptides, peptides_to_transfer.size() * sizeof(int));
    hipMemcpy(d_peptides, peptides_to_transfer.data(), peptides_to_transfer.size() * sizeof(int), hipMemcpyHostToDevice);
    std::cout << "peps allocated";

    pep_num = peptides.size();
    pep_length = peptides[0].size();
  } 
}

std::vector<int> applyScoring(const int *cache, unsigned int cache_size){

  size_t block_size = 32;
  size_t grid_size = (pep_num + 1) / block_size + 1;

  // transfer cache
  //  d_cache = d_peptides + pep_num * pep_length + 1;

  int *d_cache, *d_result;
  hipMalloc((void **)&d_cache, cache_size * sizeof(int));
  hipMemcpy(d_cache, cache, cache_size * sizeof(int), hipMemcpyHostToDevice);

  // allocate memory for result variable
  //int *d_result = d_cache + cache_size + 1;

  hipMalloc((void **)&d_result, pep_num * sizeof(int));

  // score
  std::cout << "mem allocated";
  score <<<block_size, grid_size>>>(d_peptides, d_cache, d_result, pep_length);
  
  // get results
  std::vector<int> result;
  hipMemcpy(result.data(), d_result, pep_num * sizeof(int), hipMemcpyDeviceToHost);

  std::cout << "scored";

  hipFree(d_cache);
  hipFree(d_peptides);
  hipFree(d_result);

  return result;
}
