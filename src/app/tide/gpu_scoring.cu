#include "hip/hip_runtime.h"
#include <iostream>
#include "gpu_scoring.cuh"

int *d_peptides;

size_t pep_length;
size_t pep_num;

__global__ void score(int *d_peptides, int *d_cache, int *d_result, size_t pep_length){
  
  int current_peptide = blockDim.x * blockIdx.x + threadIdx.x;
  int result = 0;
  for (size_t i = 0; i < pep_length; ++i) {
      int peak = d_peptides[current_peptide * pep_length + i];
      if(peak != -1){
        result += d_cache[peak];
      }
  }
  d_result[current_peptide] = result;

}

void transferPeptides(std::vector<std::vector<int>> peptides){

   int devices = 0; 
   hipError_t err = hipGetDeviceCount(&devices); 

   if (devices > 0 && err == hipSuccess) 
   { 
     hipSetDevice(0);
      
     size_t free, total;
     hipMemGetInfo(&free, &total);

     // Form a single vector 
     std::vector<int> peptides_to_transfer;
     for(auto pep_iter = peptides.begin(); pep_iter != peptides.end(); pep_iter++){
       copy((*pep_iter).begin(), (*pep_iter).end(), back_inserter(peptides_to_transfer));
     }

     for(int i = 0; i < 100; ++i){
	std::cout << "\nPep " << i << " : "<< peptides_to_transfer[i];
     }     

     hipError_t err = hipMalloc((void **)&d_peptides, peptides_to_transfer.size() * sizeof(int));
     hipMemcpy(d_peptides, peptides_to_transfer.data(), peptides_to_transfer.size() * sizeof(int), hipMemcpyHostToDevice);

     pep_num = peptides.size();
     pep_length = peptides[0].size();
   } 
 }

 std::vector<int> applyScoring(const int *cache, unsigned int cache_size){

   size_t block_size = 32;
   size_t grid_size = (pep_num + 1) / block_size + 1;

   // Allocate mem for cache peptides and storing results
   int *d_cache, *d_result;
   
   std::cout << "\nCACHE_SIZE: " << cache_size;
   hipError_t err = hipMalloc((void **)&d_cache, cache_size * sizeof(int));
   std::cout << "\nMALLOC CACHE: " << err;
   for(int i = 0, j = 0; i < 100; ++j){
	if(cache[j] != 0){
		std::cout << "\nCACHE " << j << " : "<< cache[j];
		++i;
	}
   }     

   // Copy cache peptides to GPU
   err = hipMemcpy(d_cache, cache, cache_size * sizeof(int), hipMemcpyHostToDevice);
   std::cout << "\nMemcpy CACHE: " << err;

   err = hipMalloc((void **)&d_result, pep_num * sizeof(int));
   std::cout << "\nMALLOC result: " << err;
   
   std::cout << "\nPEP_LEN: " << pep_length << "\nPEP_NUM: " << pep_num;
   // Run scoring function on GPU
   score <<<block_size, grid_size>>>(d_peptides, d_cache, d_result, pep_length);
  
   // Get results
   std::vector<int> result(pep_num);
   err = hipMemcpy(result.data(), d_result, pep_num * sizeof(int), hipMemcpyDeviceToHost);
   std::cout << "\nMemcpy result: " << err;

   hipFree(d_cache);
   hipFree(d_peptides);
   hipFree(d_result);

   return result;
}
