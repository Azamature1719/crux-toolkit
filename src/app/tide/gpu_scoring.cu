#include "hip/hip_runtime.h"
#include "gpu_scoring.cuh"

unsigned int *d_peptides;

void setDeviceProperties(int deviceNum, size_t warpSize, size_t spectrumMatchingOnce){
  int devices = 0; 
  hipError_t err = hipGetDeviceCount(&devices); 

  if (devices > 0 && err == hipSuccess) 
  { 
    hipSetDevice(deviceNum);
      
    // Get GPU device properties - could be used for memory configuration
    hipDeviceProp_t deviceProp; 
    hipGetDeviceProperties(&deviceProp, deviceNum);

    // Get all free memory on GPU device
    size_t free, total;
    hipMemGetInfo(&free, &total);
    hipMalloc((void **)&d_peptides, free);

    // Calculating property values
    size_t block_size = warpSize;
    size_t grid_size = (spectrumMatchingOnce + 1) / block_size + 1;
  } 
  else
  { 
    //carp(CARP_FATAL, "There are no GPU devices");
  } 
}

void transferDataToDevice(std::vector<unsigned int> peptides){
    hipMemcpy(d_peptides, peptides.data(), peptides.size() * sizeof(unsigned int), hipMemcpyHostToDevice);
}

// scoring function
__global__ void score(){
  
}
